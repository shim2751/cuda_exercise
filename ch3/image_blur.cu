#include "hip/hip_runtime.h"
#include "ch3.h"

__global__
void imageBlurKernel(unsigned char* Pin, unsigned char* Pout, int width, int height, int r){
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int ch = threadIdx.z;

    if(col < width && row < height){
        int pixelVal = 0; 
        int pixelNum = 0;
        for(int i=-r; i<r+1; i++){
            for(int j=-r; j<r+1; j++){
                int curCol = col + i;
                int curRow = row + j;
                if(0 < curCol && curCol < width && 0 < curRow && curRow < height){
                    pixelVal += Pin[(width * curRow + curCol) * CHANNEL + ch];
                    pixelNum++;
                }
            }
        }
        Pout[(width*row+col)*CHANNEL + ch] = pixelVal/pixelNum;
    }
    
}

void image_blur(unsigned char* Pin, unsigned char* Pout, int width, int height, int radius){
    unsigned char* Pin_d, *Pout_d;
    int size = width*height*sizeof(unsigned char)*CHANNEL;

    hipMalloc((void **) &Pin_d, size);
    hipMalloc((void **) &Pout_d, size);
    
    hipMemcpy(Pin_d, Pin, size, hipMemcpyHostToDevice);

    dim3 grid_dim(ceil(width/16.0), ceil(height/16.0), 1);
    dim3 block_dim(16, 16, 3);
    imageBlurKernel<<<grid_dim, block_dim>>>(Pin_d, Pout_d, width, height, radius);

    hipMemcpy(Pout, Pout_d, size, hipMemcpyDeviceToHost);

    hipFree(Pin_d);
    hipFree(Pout_d);
}