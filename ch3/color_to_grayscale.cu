#include "hip/hip_runtime.h"
#include "ch3.h"

__global__
void colorToGrayscaleKernel(unsigned char* Pin, unsigned char* Pout, int width, int height){
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if(col < width && row < height){
        int grayOffset = width*row + col;

        int rgbOffset = grayOffset * CHANNEL;
        unsigned char r = Pin[rgbOffset];
        unsigned char g = Pin[rgbOffset+1];
        unsigned char b = Pin[rgbOffset+2];
        
        Pout[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }
}

void color_to_grayscale(unsigned char* Pin, unsigned char* Pout, int width, int height){
    unsigned char* Pin_d, *Pout_d;
    int size = width*height*sizeof(unsigned char);

    hipMalloc((void **) &Pin_d, size*CHANNEL);
    hipMalloc((void **) &Pout_d, size);
    
    hipMemcpy(Pin_d, Pin, size*CHANNEL, hipMemcpyHostToDevice);

    dim3 grid_dim(ceil(width/16.0), ceil(height/16.0), 1);
    dim3 block_dim(16, 16, 1);
    colorToGrayscaleKernel<<<grid_dim, block_dim>>>(Pin_d, Pout_d, width, height);

    hipMemcpy(Pout, Pout_d, size, hipMemcpyDeviceToHost);

    hipFree(Pin_d);
    hipFree(Pout_d);
}