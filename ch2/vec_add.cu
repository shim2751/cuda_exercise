#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "vec_add.h"

__global__
void vecAddKernel(float* A, float* B, float* C, int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<n){
        C[i] = A[i] + B[i];
    }
}

void vec_add(float* A, float* B, float* C, int n){
    float *A_d, *B_d, *C_d;
    int size = n * sizeof(float);

    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
    
    vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);

    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}